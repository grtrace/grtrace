#include "hip/hip_runtime.h"
#include "basicdefs.h"
#include <math.h>

extern "C"
__global__ void vectorAdd(double *a, double *b, double *c, int n)
{
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	
	if (id < n)
	{
		c[id] = a[id] + b[id];
	}
}
